#include "hip/hip_runtime.h"

#include <vector>
#include <iostream>

#include <thrust/iterator/discard_iterator.h>
#include <thrust/reduce.h>

using namespace std;

auto main () -> int {

    auto const ones = vector<int>{1, 1, 1, 0, 0, 1, 0, 1, 1, 1, 1};
    auto       sums = vector<int>(5);

    thrust::reduce_by_key(
        ones.cbegin(),                   // keys input
        ones.cend(),   
        ones.cbegin(),                   // values input
        thrust::make_discard_iterator(), // keys output
        sums.begin());

    auto const max = *thrust::max_element(
        sums.cbegin(), sums.cend());

    std::cout << max << '\n';

}
